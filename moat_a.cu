#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <iostream>
#include <algorithm>
#include <string>
#include <cstddef>
#include <vector>
#include <sys/stat.h>
#include <errno.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "variant_permutation_v3.h"

using namespace std;

#define STRSIZE 1000
#define NUMTHREADSBASE 32

// Refactorization of the code that turns a chromosome string into an integer
__host__ int chr2int (string chr_str) {
	if (chr_str == "chrX") {
		return 23;
	} else if (chr_str == "chrY") {
		return 24;
	} else if (chr_str == "chrM" || chr_str == "chrMT") {
		return 25;
	} else {
		string chr_part = chr_str.substr(3);
		return atoi(chr_part.c_str());
	}
}

__device__ bool gpuCmpIntervals (int var_chr, int var_start, int var_end, int ann_chr, int ann_start, int ann_end) {
	if (var_chr != ann_chr) {
		return (var_chr < ann_chr);
	} else if (var_start != ann_start) {
		return (var_start < ann_start);
	} else if (var_end != ann_end) {
		return (var_end < ann_end);
	} else { // The intervals are equal, so return false since a is not less than b
		return false;
	}
}

inline void GPUassert(hipError_t code, const char * file, int line, bool Abort=true)
{
    if (code != 0) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code),file,line);
        if (Abort) exit(code);
    }       
}

#define GPUerrchk(ans) { GPUassert((ans), __FILE__, __LINE__); }

__device__ void BottomUpMerge(int* target_array, int* temp_array, int left_head, int right_head, int tail) {
	int left_ptr = left_head;
	int right_ptr = right_head;
	int end_ptr = tail;
	
	/* While there are elements in the left or right runs */
  for (int j = left_ptr; j < end_ptr; j++) {
  
  	/* If left run head exists and is <= existing right run head */
  	if (left_ptr < right_head && (right_ptr >= tail || target_array[left_ptr] <= target_array[right_ptr])) {
  		temp_array[j] = target_array[left_ptr];
  		left_ptr++;
  	} else {
  		temp_array[j] = target_array[right_ptr];
  		right_ptr++;
  	}
  }
}

__device__ void CopyArray(int* source, int* dest, int n) {
	for (int i = 0; i < n; i++) {
		dest[i] = source[i];
	}
}

__device__ void BottomUpSort(int* target_array, int n) {
	int *temp_array = (int *)malloc(n*sizeof(int));
	for (int width = 1; width < n; width = 2*width) {
		for (int i = 0; i < n; i = i+2*width) {
			int left_ptr = i;
			int right_ptr = (int)fmin((double)(i+width), (double)n);
			int end_ptr = (int)fmin((double)(i+2*width), (double)n);
			BottomUpMerge(target_array, temp_array, left_ptr, right_ptr, end_ptr);
		}
		// Copy work done in temp_array into target_array for next iteration
		CopyArray(temp_array, target_array, n);
	}
	free(temp_array);
}

__device__ void intersection_kernel(int start, int end, int* gpu_var_chr, int* gpu_var_start, int* gpu_var_end, int* gpu_ann_chr, int* gpu_ann_start, int* gpu_ann_end, int* gpu_var_arr_length, int* gpu_n, int* gpu_dmin, int* gpu_dmax, double* gpu_pvalues) {

	// DEBUG
	// int tid = threadIdx.x + blockIdx.x * blockDim.x;
	// printf("Intersection kernel %d\n", tid);

	for (int i = start; i <= end; i++) {
		// Unpack the current annotation
		int this_ann_chr = gpu_ann_chr[i];
		int this_ann_start = gpu_ann_start[i];
		int this_ann_end = gpu_ann_end[i];
		
		// Initialize the variant pointer
		int vlength = *gpu_var_arr_length;
		vlength = vlength - 1;
		int vthis = vlength/2;
		vlength = vlength/2;
		
		int vlength_const = *gpu_var_arr_length;
		
		int this_var_chr;
		int this_var_start;
		int this_var_end;
		
		// Keep track of whether the target is greater or less than the current variant
		// Also what was the last comparison
		int prev_is_greater = -1;
		int is_greater = -1;
		
		int int_variants;
		
		// return;
		// DEBUG
		// int test = 0;
		
		while (1) {
			// Unpack current variant
			this_var_chr = gpu_var_chr[vthis];
			this_var_start = gpu_var_start[vthis];
			this_var_end = gpu_var_end[vthis];
			
			// DEBUG
// 			printf("Variant %d: %d, %d, %d\n", test, this_var_chr, this_var_start, this_var_end);
// 			test++;
		
			// Check for intersection
			if (this_var_chr == this_ann_chr && this_var_start <= this_ann_end && this_ann_start <= this_var_end) {
				int_variants = 1;
				break;
			} else {
				if (vlength > 1) { // vlength does not fall below 1
					vlength = vlength/2;
				}
				if (!(gpuCmpIntervals(this_var_chr, this_var_start, this_var_end, this_ann_chr, this_ann_start, this_ann_end))) {
					if (vthis == 0) { // Don't read off the end of the array
						int_variants = 0;
						break;
					} else {
						// Take the smaller half
						vthis = vthis-vlength;
						prev_is_greater = is_greater;
						is_greater = 0;
					}
				} else {
					if (vthis == vlength_const - 1) { // Don't read off the end of the array
						int_variants = 0;
						break;
					} else {
						// Take the larger half
						vthis = vthis+vlength;
						prev_is_greater = is_greater;
						is_greater = 1;
					}
				}
				if (vlength == 1 && ((prev_is_greater == 1 && is_greater == 0) || (prev_is_greater == 0 && is_greater == 1))) { // No intersection
					int_variants = 0;
					break;
				}
			}
		}
		
		// DEBUG
		// printf("After intersection found\n");
		// printf("<-- %d, %d, %d -->\n", this_ann_chr, this_ann_start, this_ann_end);
		// printf("int_variants: %d\n", int_variants);
		
		// int int_variants = 1;
		
		int v_anchor = vthis;
		
		if (v_anchor != 0) {
			vthis--;
		
			// Unpack current variant
			this_var_chr = gpu_var_chr[vthis];
			this_var_start = gpu_var_start[vthis];
			this_var_end = gpu_var_end[vthis];
		
			// Search for intersecting variants bidirectionally
			while (this_var_chr == this_ann_chr && this_var_start <= this_ann_end && this_ann_start <= this_var_end) {
				int_variants++;
				vthis--;
				this_var_chr = gpu_var_chr[vthis];
				this_var_start = gpu_var_start[vthis];
				this_var_end = gpu_var_end[vthis];
			}
		}
		
		if (v_anchor != vlength_const-1) {
			vthis = v_anchor;
			vthis++;
		
			// Unpack current variant
			this_var_chr = gpu_var_chr[vthis];
			this_var_start = gpu_var_start[vthis];
			this_var_end = gpu_var_end[vthis];
		
			// Search for intersecting variants bidirectionally
			while (this_var_chr == this_ann_chr && this_var_start <= this_ann_end && this_ann_start <= this_var_end) {
				int_variants++;
				vthis++;
				this_var_chr = gpu_var_chr[vthis];
				this_var_start = gpu_var_start[vthis];
				this_var_end = gpu_var_end[vthis];
			}
		}
		
		// DEBUG
		// printf("Find random bins\n");
		// printf("<-- %d, %d, %d -->\n", this_ann_chr, this_ann_start, this_ann_end);
		// printf("int_variants: %d\n", int_variants);
		
		// Number of random bins to select
		int n = (*gpu_n);
	
		// The minimum distance between element and random bin
		int dmin = (*gpu_dmin);
	
		// The maximum distance between element and random bin
		int dmax = (*gpu_dmax);
		
		// Pick random bins from surrounding regions
		// We take n/2 from the upstream region, and n/2 from the downstream regions
		// Random number drawn from [0, dmax - dmin - (annotation's length)]
		// Save only the start coordinate, the chr and end can be derived JIT
		int range = dmax - dmin - (this_ann_end - this_ann_start + 1);
		int *upstream_start = (int *)malloc((n/2)*sizeof(int));
		int *downstream_start = (int *)malloc((n/2)*sizeof(int));
		
		// Upstream bin selection
		// Configure where the start of this range is
		// int rand_range_chr = this_ann_chr;
		int rand_range_start = this_ann_start - dmax;
		
		hiprandState *d_state;
		d_state = (hiprandState *)malloc(sizeof(hiprandState));
		int tid = threadIdx.x + blockIdx.x * blockDim.x;
		hiprand_init(65536, tid, 0, d_state);
		
		for (int j = 0; j < n/2; j++) {
			float this_rand = hiprand_uniform(d_state);
			int rand_start = this_rand*range;
			rand_start += rand_range_start;
			// int rand_start = rand() % range;
			
			// DEBUG
			// printf("this_rand_start: %d\n", rand_start);
			
			upstream_start[j] = rand_start;
		}
		
		// Downstream bin selection
		// Configure where the start of this range is
		// rand_range_chr = this_ann_chr;
		rand_range_start = this_ann_end + dmin;
		
		for (int j = 0; j < n/2; j++) {
			float this_rand = hiprand_uniform(d_state);
			int rand_start = this_rand*range;
			rand_start += rand_range_start;
			// int rand_start = rand() % range;
			
			// DEBUG
			// printf("this_rand_start: %d\n", rand_start);
			downstream_start[j] = rand_start;
		}
		
		// DEBUG: Check upstream and downstream random bins
// 		for (int k = 0; k < (n/2); k++) {
// 			printf("Upstream bin %d: %d\n", k, upstream_start[k]);
// 		}
// 		for (int k = 0; k < (n/2); k++) {
// 			printf("Downstream bin %d: %d\n", k, downstream_start[k]);
// 		}
		
		// Find the intersecting variants for the random bins
		
		// Sort the upstream and downstream random bins
		BottomUpSort(upstream_start, (n/2));
		BottomUpSort(downstream_start, (n/2));
		// gpuIntervalMergeSortByEnd(upstream_chr, upstream_start, upstream_end, upstream_chr_sorted, upstream_start_sorted, upstream_end_sorted, 0, (n/2)-1);
		// gpuIntervalMergeSortByStart(downstream_chr, downstream_start, downstream_end, downstream_chr_sorted, downstream_start_sorted, downstream_end_sorted, 0, (n/2)-1);
		
		// DEBUG: Check upstream and downstream random bins (sorted)
// 		for (int k = 0; k < (n/2); k++) {
// 			printf("Upstream bin (sorted) %d: %d\n", k, upstream_start[k]);
// 		}
// 		for (int k = 0; k < (n/2); k++) {
// 			printf("Downstream bin (sorted) %d: %d\n", k, downstream_start[k]);
// 		}
		
	 	// Upstream bins: search backwards from the variant at gpu_var_*[v_anchor]
	 	unsigned int vpointer2 = v_anchor;
	 	
	 	// A collection of intersecting variants counts from the random bins
	 	int *varcounts = (int *)malloc(n*sizeof(int));
	 	int varcounts_index = 0;
	 	
	 	// Backwards search!
	 	unsigned int j = (n/2);
	 	do {
	 		j--;
	 		
	 		// How many variants intersect this bin?
 			int this_variants = 0;
 			
 			// Unpack the current annotation
 			int upstream_ann_chr = this_ann_chr;
 			int upstream_ann_start = upstream_start[j];
 			int upstream_ann_end = upstream_ann_start + (this_ann_end - this_ann_start);
 			
 			// Unpack the current variant
 			int upstream_var_chr = gpu_var_chr[vpointer2];
 			int upstream_var_start = gpu_var_start[vpointer2];
 			int upstream_var_end = gpu_var_end[vpointer2];
 			
 			// Now count the intersecting variants
 			// vpointer2 points to the "earliest" possible annotation, and vpointer3
			// points to the variants up until the last intersecting with the annotation
			unsigned int vpointer3 = vpointer2;
			
			// While vpointer3 does not go past the current annotation
			while (upstream_var_chr > upstream_ann_chr || (upstream_var_chr == upstream_ann_chr && upstream_var_start >= upstream_ann_start)) {
				
				// If the current variant intersects the current annotation, increment target_variants
				if (upstream_var_chr == upstream_ann_chr && upstream_ann_start <= upstream_var_end && upstream_var_start <= upstream_ann_end) {
					this_variants++;
				} else { // Update vpointer2
					if (vpointer3 != 0) {
						vpointer2 = vpointer3 - 1;
					}
				}
				// Now update the cur_var
				if (vpointer3 == 0) {
					break;
				}
				vpointer3--;
				
				upstream_var_chr = gpu_var_chr[vpointer3];
				upstream_var_start = gpu_var_start[vpointer3];
				upstream_var_end = gpu_var_end[vpointer3];
			}
			
			// this_variants has been settled, save for output
			varcounts[varcounts_index] = this_variants;
			varcounts_index++;
		} while (j > 0);
		
		// Downstream bins: a more straight forward search :)
		vpointer2 = v_anchor;
		
		for (unsigned int j = 0; j < (n/2); j++) {
			
			// How many variants intersect this bin?
			int this_variants = 0;
			
			// Unpack the current annotation
			int downstream_ann_chr = this_ann_chr;
 			int downstream_ann_start = downstream_start[j];
 			int downstream_ann_end = downstream_ann_start + (this_ann_end - this_ann_start);
 			
 			// Unpack the current variant
 			int downstream_var_chr = gpu_var_chr[vpointer2];
 			int downstream_var_start = gpu_var_start[vpointer2];
 			int downstream_var_end = gpu_var_end[vpointer2];
 			
 			// Now count the intersecting variants
 			// vpointer2 points to the "earliest" possible variant, and vpointer3
 			// points to the variants up until the last intersecting with the annotation
 			unsigned int vpointer3 = vpointer2;
 			
 			// While vpointer3 does not go past the current annotation
 			while (downstream_var_chr < downstream_ann_chr || (downstream_var_chr == downstream_ann_chr && downstream_var_end <= downstream_ann_end)) {
 				
 				// If the current variant intersects the current annotation, increment target_variants
 				if (downstream_var_chr == downstream_ann_chr && downstream_ann_start <= downstream_var_end && downstream_var_start <= downstream_ann_end) {
 					this_variants++;
 				} else { // Update vpointer2
 					if (vpointer3 != (vlength_const)-1) {
 						vpointer2 = vpointer3 + 1;
 					}
 				}
 				// Now update the cur_var
 				if (vpointer3 == (vlength_const)-1) {
 					break;
 				}
 				vpointer3++;
 				
 				downstream_var_chr = gpu_var_chr[vpointer3];
 				downstream_var_start = gpu_var_start[vpointer3];
 				downstream_var_end = gpu_var_end[vpointer3];
 			}
 			
 			// this_variants has been settled, save for output
 			varcounts[varcounts_index] = this_variants;
 			varcounts_index++;
 		}
 		
 		// DEBUG
//  		for (int k = 0; k < n; k++) {
//  			printf("Varcounts %d: %d\n", k, varcounts[k]);
//  		}
 		
 		// P-value calculation: how many of the random bins have at least as many
 		// variants at k_t?
 		int overbins = 0;
 		for (unsigned int j = 0; j < n; j++) {
 			if (varcounts[j] >= int_variants) {
 				overbins++;
 			}
 		}
 		
 		double fraction = (double)overbins/(double)n;
 		gpu_pvalues[i] = fraction;
 		
 		// Malloc free the temp arrays
 		free(upstream_start);
 		free(downstream_start);
 		free(varcounts);
 		free(d_state);
 		
 		// DEBUG
 		// printf("GPU pvalue %d: %f\n", i, fraction);
	}
}

__global__ void apportionWork(int* gpu_var_chr, int* gpu_var_start, int* gpu_var_end, int* gpu_ann_chr, int* gpu_ann_start, int* gpu_ann_end, int* gpu_var_arr_length, int* gpu_ann_arr_length, int* gpu_n, int* gpu_dmin, int* gpu_dmax, double *gpu_pvalues) {
// __global__ void apportionWork() {

	// DEBUG
	// printf("Running thread\n");
	// *test_int_gpu = 247;
	
	// Which thread am I?
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int total_threads = NUMTHREADSBASE*NUMTHREADSBASE;
	
	// DEBUG
	// printf("%d\n", *gpu_ann_arr_length);
	// printf("%d\n", total_threads);
	
	int length = *gpu_ann_arr_length;
	
	// int kernel_annsize = (*gpu_ann_arr_length)/4;
	int kernel_annsize = length/total_threads;
	int mod = length%total_threads;
	int start;
	int end;
	
	if (kernel_annsize > 0) {
		if (tid < mod) {
			start = tid*(kernel_annsize+1);
			end = ((tid+1)*(kernel_annsize+1))-1;
		} else {
			start = (mod*(kernel_annsize+1))+((tid-mod)*kernel_annsize);
			end = ((mod*(kernel_annsize+1))+((tid-mod+1)*kernel_annsize))-1;
		}
		
		// DEBUG: Print the thread ID, start index, and end index
// 		printf("Thread ID: %d; start index: %d; end index: %d\n", tid, start, end);
// 		return;
		
		intersection_kernel(start, end, gpu_var_chr, gpu_var_start, gpu_var_end, gpu_ann_chr, gpu_ann_start, gpu_ann_end, gpu_var_arr_length, gpu_n, gpu_dmin, gpu_dmax, gpu_pvalues);
	} else {
		start = tid;
		end = tid;
		
		// DEBUG: Print the thread ID, start index, and end index
		// printf("Thread ID: %d; start index: %d; end index: %d\n", tid, start, end);
		
		if (tid < length) {
			intersection_kernel(start, end, gpu_var_chr, gpu_var_start, gpu_var_end, gpu_ann_chr, gpu_ann_start, gpu_ann_end, gpu_var_arr_length, gpu_n, gpu_dmin, gpu_dmax, gpu_pvalues);
		}
	}
}

/*
 * Subroutine that merges the input intervals (3col)
 * Assumes input interval array is sorted
 */
__host__ vector<vector<string> > merge_intervals (vector<vector<string> > starting_intervals) {
	
	// Output vector
	vector<vector<string> > resulting_intervals;
	
	for (unsigned int i = 0; i < starting_intervals.size(); i++) {
		if (i == 0) { // Just push the first interval straight onto resulting_intervals
			resulting_intervals.push_back(starting_intervals[i]);
		} else { // Compare the i'th starting to the last resulting
			vector<string> interval1 = resulting_intervals[resulting_intervals.size()-1];
			vector<string> interval2 = starting_intervals[i];
			
			vector<vector<string> > vec1;
			vec1.push_back(interval1);
			
			vector<vector<string> > int_interval = intersecting_intervals(vec1, interval2);
			
			// If there's anything in int_interval, there is an overlap, otherwise no
			if (int_interval.size() == 0) { // No intersection
				resulting_intervals.push_back(starting_intervals[i]);
			} else { // Yes intersection, create a merged interval
				vector<string> merged;
				merged.push_back(interval1[0]);
				int new_start = min(atoi(interval1[1].c_str()), atoi(interval2[1].c_str()));
				int new_end = max(atoi(interval1[2].c_str()), atoi(interval2[2].c_str()));
				
				char new_start_cstr[STRSIZE];
				sprintf(new_start_cstr, "%d", new_start);
				merged.push_back(string(new_start_cstr));
		
				char new_end_cstr[STRSIZE];
				sprintf(new_end_cstr, "%d", new_end);
				merged.push_back(string(new_end_cstr));
				
				resulting_intervals.pop_back();
				resulting_intervals.push_back(merged);
			}
		}
	}
	return resulting_intervals;
}

/* 
 * This code takes as input a variant track and an annotation track. For each
 * element, count intersecting variants, and select n (hardcoded) random bins
 * from a range upstream and downstream of the element. These random bins will
 * fall within d_min and d_max distance from the element, and the n bins will
 * be split evenly between upstream and downstream bins. Then, calculate a p-value
 * for mutation burden based on how many of these random bins have more intersecting
 * variants.
 */
 
int main (int argc, char* argv[]) {
	
	/* User-supplied arguments */
	
	// Number of random bins to select
	int n;
	
	// The minimum distance between element and random bin
	int dmin;
	
	// The maximum distance between element and random bin
	int dmax;
	
	// File with prohibited coordinates
	// Expected format: tab(chr, start, end, ...)
	string prohibited_file;
	
	// File with single nucleotide variants
	// Expected format: tab(chr, start, end, ...)
	string vfile;
	
	// File with annotations to study for mutation burden
	// Expected format: tab(chr, start, end, name, ...)
	string afile;
	
	// File with the output
	// Format: tab(chr, start, end, name, p-value)
	string outfile;
	
	if (argc != 8) {
		fprintf(stderr, "Usage: moat_a_gpu [# of permutations] [d_min] [d_max] [prohibited regions file] [variant file] [annotation file] [output file]. Exiting.\n");
		return 1;
	} else {
		n = atoi(argv[1]);
		dmin = atoi(argv[2]);
		dmax = atoi(argv[3]);
		prohibited_file = string(argv[4]);
		vfile = string(argv[5]);
		afile = string(argv[6]);
		outfile = string(argv[7]);
	}
	
	// Verify files, and import data to memory
	struct stat vbuf;
	if (stat(vfile.c_str(), &vbuf)) { // Report the error and exit
		fprintf(stderr, "Error trying to stat %s: %s\n", vfile.c_str(), strerror(errno));
		return 1;
	}
	// Check that the file is not empty
	if (vbuf.st_size == 0) {
		fprintf(stderr, "Error: Variant file cannot be empty. Exiting.\n");
		return 1;
	}
	
	struct stat abuf;
	if (stat(afile.c_str(), &abuf)) { // Report the error and exit
		fprintf(stderr, "Error trying to stat %s: %s\n", afile.c_str(), strerror(errno));
		return 1;
	}
	// Check that the file is not empty
	if (abuf.st_size == 0) {
		fprintf(stderr, "Error: Annotation file cannot be empty. Exiting.\n");
		return 1;
	}
	
	struct stat pbuf;
	if (stat(prohibited_file.c_str(), &pbuf)) { // Report the error and exit
		fprintf(stderr, "Error trying to stat %s: %s\n", prohibited_file.c_str(), strerror(errno));
		return 1;
	}
	// Check that the file is not empty
	if (pbuf.st_size == 0) {
		fprintf(stderr, "Error: Prohibited regions file cannot be empty. Exiting.\n");
		return 1;
	}
	
	/* Data structures for the starting data */
	// Variant arrays, contains variants of the format vector(chr, start, end)
	vector<vector<string> > var_array;
	
	// Annotation arrays, contains annotations of the format vector(chr, start, end, name)
	vector<vector<string> > ann_array;
	
	// Prohibited regions array, contains annotations of the format vector(chr, start, end)
	vector<vector<string> > prohibited_regions;
	
	// DEBUG
	// printf("Breakpoint 1\n");
	
	// Bring variant file data into memory
	// Save the first 3 columns, ignore the rest if there are any
	char linebuf[STRSIZE];
	FILE *vfile_ptr = fopen(vfile.c_str(), "r");
	while (fgets(linebuf, STRSIZE, vfile_ptr) != NULL) {
		string line = string(linebuf);
		
		// DEBUG
		// printf("%s\n", line.c_str());
		
		// Extract chromosome, start, and end from line (first 3 columns)
		vector<string> vec;
		for (int i = 0; i < 3; i++) {
			size_t ws_index = line.find_first_of("\t\n");
			string in = line.substr(0, ws_index);
			vec.push_back(in);
			line = line.substr(ws_index+1);
		}
		
		// If this is not a standard chromosome, then remove this row
		if (chr2int(vec[0]) == 0) {
			continue;
		}
		
		var_array.push_back(vec);
	}
	// Check feof of vfile
	if (feof(vfile_ptr)) { // We're good
		fclose(vfile_ptr);
	} else { // It's an error
		char errstring[STRSIZE];
		sprintf(errstring, "Error reading from %s", vfile.c_str());
		perror(errstring);
		return 1;
	}
	
	// Bring annotation file data into memory
	FILE *afile_ptr = fopen(afile.c_str(), "r");
	while (fgets(linebuf, STRSIZE, afile_ptr) != NULL) {
		string line = string(linebuf);
		
		// DEBUG
		// printf("%s", line.c_str());
		
		// Extract chromosome, start, end, and name from line (first 4 columns)
		vector<string> vec;
		for (int i = 0; i < 4; i++) {
			size_t ws_index = line.find_first_of("\t\n");
			string in = line.substr(0, ws_index);
			vec.push_back(in);
			line = line.substr(ws_index+1);
		}
		
		// If this is not a standard chromosome, then remove this row
		if (chr2int(vec[0]) == 0) {
			continue;
		}
		
		ann_array.push_back(vec);
	}
	// Check feof of vfile
	if (feof(afile_ptr)) { // We're good
		fclose(afile_ptr);
	} else { // It's an error
		char errstring[STRSIZE];
		sprintf(errstring, "Error reading from %s", afile.c_str());
		perror(errstring);
		return 1;
	}
	
	// Import prohibited regions file
	FILE *prohibited_file_ptr = fopen(prohibited_file.c_str(), "r");
	while (fgets(linebuf, STRSIZE, prohibited_file_ptr) != NULL) {
	
		string line = string(linebuf);
		
		// Extract chromosome, start, and end from line (first 3 columns)
		vector<string> vec;
		for (int i = 0; i < 3; i++) {
			size_t ws_index = line.find_first_of("\t\n");
			string in = line.substr(0, ws_index);
			vec.push_back(in);
			line = line.substr(ws_index+1);
		}
		
		// If this is not a standard chromosome, then remove this row
		if (chr2int(vec[0]) == 0) {
			continue;
		}
		
		prohibited_regions.push_back(vec);
	}
	// Check feof of prohibited_file_ptr
	if (feof(prohibited_file_ptr)) { // We're good
		fclose(prohibited_file_ptr);
	} else { // It's an error
		char errstring[STRSIZE];
		sprintf(errstring, "Error reading from %s", prohibited_file.c_str());
		perror(errstring);
		return 1;
	}
	
	// DEBUG
	// printf("Breakpoint 2\n");
	
	// Sort the arrays
	sort(var_array.begin(), var_array.end(), cmpIntervals);
	sort(ann_array.begin(), ann_array.end(), cmpIntervals);
	sort(prohibited_regions.begin(), prohibited_regions.end(), cmpIntervals);
	
	// Merge prohibited regions
	prohibited_regions = merge_intervals(prohibited_regions);
	
	// Remove variants and annotations that intersect the blacklist regions
// 	vector<vector<string> > var_array_new;
	for (unsigned int i = 0; i < var_array.size(); i++) {
		vector<vector<string> > inter = intersecting_intervals(prohibited_regions, var_array[i]);
		if (inter.size() > 0) {
			// var_array_new.push_back(var_array[i]);
			var_array[i][0] = "chrNo";
		}
	}
// 	var_array = var_array_new;

	sort(var_array.begin(), var_array.end(), cmpIntervals);
	
	// Remove those marked for deletion
	while (var_array[var_array.size()-1][0] == "chrNo") {
		var_array.erase(var_array.end());
	}
	
	// vector<vector<string> > ann_array_new;
	for (unsigned int i = 0; i < ann_array.size(); i++) {
		vector<vector<string> > inter = intersecting_intervals(prohibited_regions, ann_array[i]);
		if (inter.size() > 0) {
			// ann_array_new.push_back(ann_array[i]);
			ann_array[i][0] = "chrNo";
		}
	}
	// ann_array = ann_array_new;
	
	sort(ann_array.begin(), ann_array.end(), cmpIntervals);
	
	// Remove those marked for deletion
	while (ann_array[ann_array.size()-1][0] == "chrNo") {
		ann_array.erase(ann_array.end());
	}
	
	// DEBUG
// 	printf("DEBUG: Sorted var_array\n");
// 	for (unsigned int i = 0; i < var_array.size(); i++) {
// 		printf("%s, %s, %s\n", var_array[i][0].c_str(), var_array[i][1].c_str(), var_array[i][2].c_str());
// 	}
	// printf("Breakpoint 3\n");
	
	// Variables for main loop
	// unsigned int var_pointer = 0;
	
	// Can malloc free the prohibited regions
	prohibited_regions.clear();
	
	// Length of each variant array
	int var_arr_length = var_array.size();
	
	// Turn the vectors into int arrays for CUDA
	// Var array
	int *var_chr = (int*)malloc(var_arr_length*sizeof(int));
	int *var_start = (int*)malloc(var_arr_length*sizeof(int));
	int *var_end = (int*)malloc(var_arr_length*sizeof(int));
	
	// Lengths of each annotation array
	int ann_arr_length = ann_array.size();
	
	// Ann array
	int *ann_chr = (int*)malloc(ann_arr_length*sizeof(int));
	int *ann_start = (int*)malloc(ann_arr_length*sizeof(int));
	int *ann_end = (int*)malloc(ann_arr_length*sizeof(int));
	
	// Variant array processing
	for (unsigned int i = 0; i < var_array.size(); i++) {
		// Unpack the current variant
		string cur_var_chr = var_array[i][0];
		string cur_var_start = var_array[i][1];
		string cur_var_end = var_array[i][2];
		
		int cur_var_chr_int;
		if (cur_var_chr == "chrX") {
			cur_var_chr_int = 24;
		} else if (cur_var_chr == "chrY") {
			cur_var_chr_int = 25;
		} else if (cur_var_chr == "chrM") {
			cur_var_chr_int = 26;
		} else {
			string cur_var_chr_part = cur_var_chr.substr(3);
			cur_var_chr_int = atoi(cur_var_chr_part.c_str());
		}
		
		// var_arr_length++;
		
		// var_chr = (int*)realloc(var_chr, var_arr_length*sizeof(int));
		var_chr[i] = cur_var_chr_int;
		
		// var_start = (int*)realloc(var_start, var_arr_length*sizeof(int));
		var_start[i] = atoi(cur_var_start.c_str());
		
		// var_end = (int*)realloc(var_end, var_arr_length*sizeof(int));
		var_end[i] = atoi(cur_var_end.c_str());
	}
	
	// Can malloc free the variant array
	var_array.clear();
	
	// Annotation array processing
	for (unsigned int i = 0; i < ann_array.size(); i++) {
		// Unpack the current variant
		string cur_ann_chr = ann_array[i][0];
		string cur_ann_start = ann_array[i][1];
		string cur_ann_end = ann_array[i][2];
		
		int cur_ann_chr_int;
		if (cur_ann_chr == "chrX") {
			cur_ann_chr_int = 24;
		} else if (cur_ann_chr == "chrY") {
			cur_ann_chr_int = 25;
		} else if (cur_ann_chr == "chrM") {
			cur_ann_chr_int = 26;
		} else {
			string cur_ann_chr_part = cur_ann_chr.substr(3);
			cur_ann_chr_int = atoi(cur_ann_chr_part.c_str());
		}
		
		// ann_arr_length++;
		
		// ann_chr = (int*)realloc(ann_chr, ann_arr_length*sizeof(int));
		ann_chr[i] = cur_ann_chr_int;
		
		// ann_start = (int*)realloc(ann_start, ann_arr_length*sizeof(int));
		ann_start[i] = atoi(cur_ann_start.c_str());
		
		// ann_end = (int*)realloc(ann_end, ann_arr_length*sizeof(int));
		ann_end[i] = atoi(cur_ann_end.c_str());
	}
	
	// Can't malloc free the annotation array
	// ann_array.clear();
	
	// DEBUG
	// printf("Breakpoint 4\n");
	
	// Begin the CUDA magic
	int *gpu_var_chr;
	int *gpu_var_start;
	int *gpu_var_end;
	
	int *gpu_ann_chr;
	int *gpu_ann_start;
	int *gpu_ann_end;
	
	int *gpu_var_arr_length;
	int *gpu_ann_arr_length;
	
	int *gpu_n;
	int *gpu_dmin;
	int *gpu_dmax;
	
	// DEBUG
// 	printf("Begin CUDA code\n");
// 	int *test_int_cpu;
// 	int *test_int_gpu;
// 	test_int_cpu = (int*)malloc(sizeof(int));
// 	hipMalloc((void**)&test_int_gpu, sizeof(int));
// 	*test_int_cpu = 246;
// 	hipMemcpy(test_int_gpu, test_int_cpu, sizeof(int), hipMemcpyHostToDevice);
	
	hipMalloc((void**)&gpu_var_chr, var_arr_length*sizeof(int));
	// GPUerrchk(hipPeekAtLastError());
	hipMalloc((void**)&gpu_var_start, var_arr_length*sizeof(int));
	// GPUerrchk(hipPeekAtLastError());
	hipMalloc((void**)&gpu_var_end, var_arr_length*sizeof(int));
	// GPUerrchk(hipPeekAtLastError());
	
	hipMalloc((void**)&gpu_ann_chr, ann_arr_length*sizeof(int));
	// GPUerrchk(hipPeekAtLastError());
	hipMalloc((void**)&gpu_ann_start, ann_arr_length*sizeof(int));
	// GPUerrchk(hipPeekAtLastError());
	hipMalloc((void**)&gpu_ann_end, ann_arr_length*sizeof(int));
	// GPUerrchk(hipPeekAtLastError());
	
	hipMalloc((void**)&gpu_var_arr_length, sizeof(int));
	// GPUerrchk(hipPeekAtLastError());
	hipMalloc((void**)&gpu_ann_arr_length, sizeof(int));
	// GPUerrchk(hipPeekAtLastError());
	
	hipMalloc((void**)&gpu_n, sizeof(int));
	hipMalloc((void**)&gpu_dmin, sizeof(int));
	hipMalloc((void**)&gpu_dmax, sizeof(int));
	
	double *gpu_pvalues;
	hipMalloc((void**)&gpu_pvalues, ann_arr_length*sizeof(double));
	// GPUerrchk(hipPeekAtLastError());
	
	hipMemcpy(gpu_var_chr, var_chr, var_arr_length*sizeof(int), hipMemcpyHostToDevice);
	// GPUerrchk(hipPeekAtLastError());
	hipMemcpy(gpu_var_start, var_start, var_arr_length*sizeof(int), hipMemcpyHostToDevice);
	// GPUerrchk(hipPeekAtLastError());
	hipMemcpy(gpu_var_end, var_end, var_arr_length*sizeof(int), hipMemcpyHostToDevice);
	// GPUerrchk(hipPeekAtLastError());

	hipMemcpy(gpu_ann_chr, ann_chr, ann_arr_length*sizeof(int), hipMemcpyHostToDevice);
	// GPUerrchk(hipPeekAtLastError());
	hipMemcpy(gpu_ann_start, ann_start, ann_arr_length*sizeof(int), hipMemcpyHostToDevice);
	// GPUerrchk(hipPeekAtLastError());
	hipMemcpy(gpu_ann_end, ann_end, ann_arr_length*sizeof(int), hipMemcpyHostToDevice);
	// GPUerrchk(hipPeekAtLastError());
	
	hipMemcpy(gpu_var_arr_length, &var_arr_length, sizeof(int), hipMemcpyHostToDevice);
	// GPUerrchk(hipPeekAtLastError());
	hipMemcpy(gpu_ann_arr_length, &ann_arr_length, sizeof(int), hipMemcpyHostToDevice);
	// GPUerrchk(hipPeekAtLastError());
	
	hipMemcpy(gpu_n, &n, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(gpu_dmin, &dmin, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(gpu_dmax, &dmax, sizeof(int), hipMemcpyHostToDevice);
	
	// Try out 16x16 and see how that goes
	int num_blocks = NUMTHREADSBASE;
	int threads_per_block = NUMTHREADSBASE;
	
	// DEBUG
	// printf("This is debug print 1\n");
	
	// DEBUG
	// printf("Breakpoint 5\n");
	
	// Adjust the heap size based on the size of the dataset
// 	if (ann_array.size() > 3000) {
// 		int fold = ann_array.size()/3000;
// 		int new_heap_size = fold*8000000;
// 		int new_heap_size = 800000000;
//  		hipDeviceSetLimit(hipLimitMallocHeapSize, new_heap_size);
// 	}
	
	apportionWork<<<num_blocks, threads_per_block>>>(gpu_var_chr, gpu_var_start, gpu_var_end, gpu_ann_chr, gpu_ann_start, gpu_ann_end, gpu_var_arr_length, gpu_ann_arr_length, gpu_n, gpu_dmin, gpu_dmax, gpu_pvalues);
	GPUerrchk(hipPeekAtLastError());
	// apportionWork<<<1,1>>>();
	
	// DEBUG
// 	printf("This is debug print 2\n");
// 	hipMemcpy(test_int_cpu, test_int_gpu, sizeof(int), hipMemcpyDeviceToHost);
// 	printf("Test int: %d\n", *test_int_cpu);

	// DEBUG
	// printf("Breakpoint 6\n");
	
	// GPUerrchk(hipDeviceSynchronize());
	
	// Collect the output values, will end with same size as ann_array
	double *pvalues = (double *)malloc(ann_array.size()*sizeof(double));
	// int block = 1000;
	
	hipMemcpy(pvalues, gpu_pvalues, ann_arr_length*sizeof(double), hipMemcpyDeviceToHost);
	// GPUerrchk(hipPeekAtLastError());
//  	if (gpu_pvalues == NULL) {
//  		printf("Malloc fail!\n");
//  	}
// 	return 0;
// 	
// 	if (ann_arr_length < block) {
// 		hipMemcpy(pvalues, gpu_pvalues, ann_arr_length*sizeof(double), hipMemcpyDeviceToHost);
// 		GPUerrchk(hipPeekAtLastError());
// 	} else {
// 		double *pvalues_ptr = pvalues;
// 		double *gpu_pvalues_ptr = gpu_pvalues;
// 		for (int k = 0; k < ann_arr_length; k += block) {
// 		
// 			// DEBUG
// 			printf("k: %d\n", k);
// 		
// 			int copyblock;
// 			if (k < ann_arr_length-block) {
// 				copyblock = ann_arr_length-k;
// 			} else {
// 				copyblock = block;
// 			}
// 			hipMemcpy(pvalues_ptr, gpu_pvalues_ptr, copyblock*sizeof(double), hipMemcpyDeviceToHost);
// 			GPUerrchk(hipPeekAtLastError());
// 			pvalues_ptr += block*sizeof(double);
// 			gpu_pvalues_ptr += block*sizeof(double);
// 		}
// 	}
	// GPUerrchk(hipDeviceSynchronize());
	
	// DEBUG
	// printf("Passed CUDA memcpy\n");
// 	for (int i = 0; i < ann_arr_length; i++) {
// 		printf("Pvalue %d: %f\n", i, gpu_pvalues[i]);
// 	}

	// DEBUG
	// printf("Breakpoint 7\n");

	// Output generation
	FILE *outfile_ptr = fopen(outfile.c_str(), "w");
	for (unsigned int i = 0; i < ann_array.size(); i++) {
		
		// Unpack the annotation
		string cur_ann_chr = ann_array[i][0];
		string cur_ann_start = ann_array[i][1];
		string cur_ann_end = ann_array[i][2];
		string cur_ann_name = ann_array[i][3];
		
		// Print the output line
		fprintf(outfile_ptr, "%s\t%s\t%s\t%s\t%f\n", cur_ann_chr.c_str(), cur_ann_start.c_str(), cur_ann_end.c_str(), cur_ann_name.c_str(), pvalues[i]);
	}
	fclose(outfile_ptr);
	
 	return 0;
}
